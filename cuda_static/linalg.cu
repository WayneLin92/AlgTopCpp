#include "hip/hip_runtime.h"
#include "mycuda.h"
#include "mycuda_public.h"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>


namespace cuda {

    /********** Reduce the matrix `m` over F_2 by the `i`th row **********/
    __global__ void EchelonKernel(int* m, int nRows, int nColumns, int i, int j) {
        extern __shared__ int sdata[];
        int tid = threadIdx.x;
        int i1 = i + tid + 1;
        int j1 = blockIdx.x;
        if (tid == 0)
            sdata[0] = m[i * nColumns + j];
        int m_i1j = m[i1 * nColumns + j];
        __syncthreads();

        while (i1 < nRows) {
            m[i1 * nColumns + j1] ^= m[i * nColumns + j1] & sdata[0] & m_i1j;
            i1 += blockDim.x;
        }
    }

    /* Wrapper for the lauch of the kernel */
    void Echelon(cuda::ArrayInt& dev_m, int nRows, int nColumns, int i, int j)
    {
        int threads = nRows - i - 1;
        if (!threads)
            return;
        int blocks = nColumns;
        if (threads > MAX_THREADS)
            threads = MAX_THREADS;
        dim3 dimBlock(threads, 1, 1);
        dim3 dimGrid(blocks, 1, 1);
        EchelonKernel <<<dimGrid, dimBlock, sizeof(int)>>> (dev_m.data(), nRows, nColumns, i, j);
    }

    /********** replace the entries of the matrix `m` with indices for compression **********/
    __global__ void ReplaceWithIndicesKernel(int* m, int index, int size) {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        if (i < size)
            m[i] = m[i] ? index + i : -1;
    }

    /* Wrapper for the lauch of the kernel */
    void ReplaceWithIndices(int* m, int index, int size)
    {
        int numBlocks;
        int numThreads;
        if (size < MAX_THREADS) {
            numBlocks = 1;
            numThreads = size;
        }
        else {
            numBlocks = (size + MAX_THREADS - 1) / MAX_THREADS;
            numThreads = MAX_THREADS;
        }
        ReplaceWithIndicesKernel<<<numBlocks, numThreads>>>(m, index, size);
#ifdef _DEBUG
        cuda::CheckLastError();
        cuda::DeviceSynchronize();
#endif
    }

    struct is_nonnegative
    {
        __host__ __device__ bool operator()(const int x) {
            return x >= 0;
        }
    };

    /* Return the row echelon form
    ** the input and out matrices are both in the form of compressed sparse rows.
    */
    array2d EchelonCuda(const array2d& matrix_csr)
    {
        /* Create the sparse matrix dev_m in GPU. */
        int nRows = (int)matrix_csr.size();
        int nColumns = 0;
        std::vector<cuda::ArrayInt> dev_rows(nRows);
        for (int i = 0; i < nRows; ++i) {
            dev_rows[i].init(matrix_csr[i]);
            if (!matrix_csr[i].empty() && nColumns < matrix_csr[i].back() + 1)
                nColumns = matrix_csr[i].back() + 1;
        }
        if (nRows == 0 || nColumns == 0)
            return array2d{};
        int size_m = nRows * nColumns;
        cuda::ArrayInt dev_m(size_m);
        cuda::FillZero(dev_m.data(), size_m * sizeof(int));
        for (int i = 0; i < (int)matrix_csr.size(); ++i) {
            int size_rowi = (int)matrix_csr[i].size();
            cuda::Decompress(dev_rows[i].data(), dev_m.data() + i * nColumns, size_rowi);
#ifdef _DEBUG
            cuda::CheckLastError();
            cuda::DeviceSynchronize();
#endif
        }

        /* Reduce the rows */
        array2d result;
        cuda::ArrayInt tmp(nColumns);
        for (int i = 0; i < nRows; ++i) {
            int index = cuda::MinIndex(dev_m.data() + i * nColumns, nColumns);
            if (index != INT_MAX) {
                Echelon(dev_m, nRows, nColumns, i, index);
#ifdef _DEBUG
                cuda::CheckLastError();
                cuda::DeviceSynchronize();
#endif
                int size_rowi = cuda::Sum(dev_m.data() + i * nColumns, nColumns);
                result.emplace_back(size_rowi);
                ReplaceWithIndices(dev_m.data() + i * nColumns + index, index, nColumns - index);
#ifdef _DEBUG
                cuda::CheckLastError();
                cuda::DeviceSynchronize();
#endif
                thrust::device_ptr<int> thrust_m = thrust::device_pointer_cast(dev_m.data());
                thrust::copy_if(thrust_m + i * nColumns, thrust_m + (i + 1) * nColumns, tmp.data(), is_nonnegative());
                cuda::Memcpy(result.back(), tmp, result.back().size());
            }
        }
        
        return result;
    }
}