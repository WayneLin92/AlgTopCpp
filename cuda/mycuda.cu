#include "mycuda.h"
#include <iostream>

void cuda::SetDevice(int device) {
    if (hipSetDevice(device) != hipSuccess) {
        std::cerr << "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?";
        throw "e31e20a";
    }
}

void cuda::Malloc(void** devPtr, size_t size)
{
    CheckLastError();
    if (hipMalloc(devPtr, size) != hipSuccess) {
        std::cerr << "hipMalloc failed!";
        throw "1e32d383";
    }
}

void cuda::Memcpy(void* dst, const void* src, size_t count, hipMemcpyKind kind)
{
    if (hipMemcpy(dst, src, count, kind) != hipSuccess) {
        std::cerr << "hipMemcpy failed!";
        throw "798e95c";
    }
}

void cuda::CheckLastError() {
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "Cuda failed: " << hipGetErrorString(cudaStatus) << '\n';
        throw "e502018d";
    }
}

void cuda::DeviceSynchronize()
{
    hipError_t cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceSynchronize returned error code " << cudaStatus << '\n';
        throw "19149f41";
    }
}