#include "mycuda.h"
#include <iostream>

void cuda::SetDevice(int device) {
    if (hipSetDevice(device) != hipSuccess) {
        std::cerr << "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?";
        throw "e31e20a";
    }
}

void cuda::Malloc(void** devPtr, size_t bytes)
{
    hipError_t cudaStatus = hipMalloc(devPtr, bytes);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << '\n';
        throw "1e32d383";
    }
}

void cuda::FillZero(void* devPtr, size_t bytes)
{
    hipError_t cudaStatus = hipMemset(devPtr, 0, bytes);
    if (cudaStatus != hipSuccess) {
        std::cerr << "FillZero failed!: " << hipGetErrorString(cudaStatus) << '\n';
        throw "798e95c";
    }
}

void cuda::Memcpy(void* dst, const void* src, size_t bytes, hipMemcpyKind kind)
{
    hipError_t cudaStatus = hipMemcpy(dst, src, bytes, kind);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpy failed: " << hipGetErrorString(cudaStatus) << '\n';
        throw "ac621fd2";
    }
}

void cuda::CheckLastError() {
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "Cuda failed: " << hipGetErrorString(cudaStatus) << '\n';
        throw "e502018d";
    }
}

void cuda::DeviceSynchronize()
{
    hipError_t cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceSynchronize failed: " << hipGetErrorString(cudaStatus) << '\n';
        throw "19149f41";
    }
}