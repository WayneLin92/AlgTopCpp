#include "hip/hip_runtime.h"
#include "mycuda.h"
#include "mycuda_public.h"
#include <hip/hip_runtime.h>
#include <>

constexpr int WARP_SIZE = 32;

/* perform first level of reduction */
template <unsigned int blockSize>
__global__ void ReduceMinKernel(const int* __restrict__ g_idata, int* __restrict__ g_odata, unsigned int n)
{
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int gridSize2 = blockSize * gridDim.x * 2;

    /* we reduce multiple elements per thread.  The number is determined by gridSize.
    ** More blocks will result in a larger gridSize and therefore fewer elements per thread */
    unsigned int i = blockIdx.x * blockSize * 2 + threadIdx.x;
    int myMin = MAX_INT;
    while (i < n) {
        myMin = min(myMin, g_idata[i]);
        if ((i + blockSize) < n)
            myMin = min(myMin, g_idata[i + blockSize]);
        i += gridSize2;
    }

    /* Reduce within warp using __shfl_down_sync */
    for (int offset = min(blockSize, WARP_SIZE) / 2; offset > 0; offset /= 2)
        myMin = min(myMin, __shfl_down_sync(0xffffffff, myMin, offset));
    if ((tid % WARP_SIZE) == 0) /* each warp puts its local sum into shared memory */
        sdata[tid / WARP_SIZE] = myMin;

    __syncthreads();

    /* Reduce shared memory using __shfl_down_sync  */
    const unsigned int size_share_memory = (blockSize / WARP_SIZE) > 0 ? (blockSize / WARP_SIZE) : 1; /* size_share_memory <= 1024/32=32 */
    const unsigned int mask_ballot = __ballot_sync(0xffffffff, tid < size_share_memory);
    if (tid < size_share_memory) {
        myMin = sdata[tid];
        for (int offset = size_share_memory / 2; offset > 0; offset /= 2)
            myMin = min(myMin, __shfl_down_sync(mask_ballot, myMin, offset));
    }

    /* write result for this block to global mem */
    if (tid == 0)
        g_odata[blockIdx.x] = myMin;
}

/* Wrapper for the lauch of the kernel
** Reduce the array `dev_in` of size `size` to the array `dev_out` of size `blocks` */
void ReduceMin(int size, int threads, int blocks, const int* dev_in, int* dev_out)
{
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);
    int smemSize = ((threads / WARP_SIZE) + 1) * sizeof(int);
    switch (threads) {
    case 1024:
        ReduceMinKernel<1024><<<dimGrid, dimBlock, smemSize>>>(dev_in, dev_out, size);
        break;
    case 512:
        ReduceMinKernel<512><<<dimGrid, dimBlock, smemSize>>>(dev_in, dev_out, size);
        break;
    case 256:
        ReduceMinKernel<256><<<dimGrid, dimBlock, smemSize>>>(dev_in, dev_out, size);
        break;
    case 128:
        ReduceMinKernel<128><<<dimGrid, dimBlock, smemSize>>>(dev_in, dev_out, size);
        break;
    case 64:
        ReduceMinKernel<64><<<dimGrid, dimBlock, smemSize>>>(dev_in, dev_out, size);
        break;
    case 32:
        ReduceMinKernel<32><<<dimGrid, dimBlock, smemSize>>>(dev_in, dev_out, size);
        break;
    case 16:
        ReduceMinKernel<16><<<dimGrid, dimBlock, smemSize>>>(dev_in, dev_out, size);
        break;
    case  8:
        ReduceMinKernel<8><<<dimGrid, dimBlock, smemSize>>>(dev_in, dev_out, size);
        break;
    case  4:
        ReduceMinKernel<4><<<dimGrid, dimBlock, smemSize>>>(dev_in, dev_out, size);
        break;
    case  2:
        ReduceMinKernel<2><<<dimGrid, dimBlock, smemSize>>>(dev_in, dev_out, size);
        break;
    case  1:
        ReduceMinKernel<1><<<dimGrid, dimBlock, smemSize>>>(dev_in, dev_out, size);
        break;
    }
}

void cuda::Min(const int* dev_in, int* dev_out, size_t size_in)
{
    int maxThreads = 256;
    int maxBlocks = 64;
    int numBlocks = 0;
    int numThreads = 0;
    getNumBlocksAndThreads((int)size_in, maxBlocks, maxThreads, numBlocks, numThreads);
    ArrayInt dev_c(numBlocks), dev_tmp(numBlocks);
    ReduceMin((int)size_in, numThreads, numBlocks, dev_in, dev_c.data());
#ifdef _DEBUG
    CheckLastError();
    DeviceSynchronize();
#endif

    int s = numBlocks;
    while (s > 1) {
        int threads = 0, blocks = 0;
        getNumBlocksAndThreads(s, maxBlocks, maxThreads, blocks, threads);
        Memcpy(dev_tmp, dev_c, s);
        ReduceMin(s, threads, blocks, dev_tmp.data(), dev_c.data());
#ifdef _DEBUG
        CheckLastError();
        DeviceSynchronize();
#endif
        s = (s + (threads * 2 - 1)) / (threads * 2);
    }
    Memcpy(dev_out, dev_c.data(), sizeof(int), hipMemcpyDeviceToDevice);
}
